
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

__global__ void hello() {
    int tid = threadIdx.x + blockIdx.x * blockDim.x; 
    printf("Hello world, thread:%d\n", tid);
}




int main(int argc, char* argv[]) {

    hipSetDevice(0);

    hello<<<3, 10>>>();

    //cudaDeviceReset();
    hipDeviceSynchronize();

    return 0;
}
